#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void addInt4Kernel( int4* a,  int4* b, int4* result) {
    // Since we are using only 1 block and 1 thread, no need for indexing
    //result[0].x = a[0].x + b[0].x;
    //result[0].y = a[0].y + b[0].y;
    //result[0].z = a[0].z + b[0].z;
    //result[0].w = a[0].w + b[0].w;
    *result = *a + *b;
}

int main() {
    // Declare and initialize host-side int4 variables
    int4 h_a = {1, 2, 3, 4};
    int4 h_b = {5, 6, 7, 8};
    int4 h_result;

    // Declare device-side int4 pointers
    int4 *d_a, *d_b, *d_result;

    // Allocate memory on the device
    hipMalloc((void**)&d_a, sizeof(int4));
    hipMalloc((void**)&d_b, sizeof(int4));
    hipMalloc((void**)&d_result, sizeof(int4));

    // Copy input data from host to device
    hipMemcpy(d_a, &h_a, sizeof(int4), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &h_b, sizeof(int4), hipMemcpyHostToDevice);

    // Launch kernel with 1 block and 1 thread
    addInt4Kernel<<<1, 1>>>(d_a, d_b, d_result);

    // Copy result back from device to host
    hipMemcpy(&h_result, d_result, sizeof(int4), hipMemcpyDeviceToHost);

    // Print the result
    printf("Result: (%d, %d, %d, %d)\n", h_result.x, h_result.y, h_result.z, h_result.w);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);

    return 0;
}

