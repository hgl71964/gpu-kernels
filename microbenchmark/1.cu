//
// 1.cu
// to ptx: nvcc -ptx 1.cu -o 1.ptx
// to run: nvcc -arch sm_80 1.cu 
// to ptx -> sass: ptxas -v -o 1.cubin --gpu-name sm_80 1.ptx
//

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int* a, int*b, int*c, long long int * clock_prof){
    int a1 = a[0];
    int b1 = b[0];
    int a2 = a[1];
    int b2 = b[1];
    int a3 = a[2];
    int b3 = b[2];

    long long int before = clock64();
    int c1 = a1 + b1;
    int c2 = a2 + b2;
    int c3 = a3 + b3;
    long long int after = clock64();

    c[0] = c1;
    c[1] = c2;
    c[2] = c3;

    *clock_prof = (after - before);
}

int main(){
    // alloc
    int *ha = (int *)malloc(sizeof(int)*100);
    int *hb = (int *)malloc(sizeof(int)*100);
    int *hc = (int *)malloc(sizeof(int)*100);
    long long int  *clock = (long long int  *)malloc(sizeof(long long int )*1);
    int *da;
    int *db;
    int *dc;
    long long int  *dclock;
    hipMalloc((void**)&db, sizeof(int)*100);
    hipMalloc((void **)&da, sizeof(int)*100);
    hipMalloc((void**)&dc, sizeof(int)*100);
    hipMalloc((void**)&dclock, sizeof(long long int )*1);
    for (int i =0;i<100;++i) {
        ha[i] = 1;
        hb[i] = 1;
        hc[i] = 0;
    }
    hipMemcpy(da, ha, sizeof(int)*100,hipMemcpyHostToDevice);
    hipMemcpy(db, hb, sizeof(int)*100, hipMemcpyHostToDevice);
    hipMemcpy(dc, hc, sizeof(int)*100, hipMemcpyHostToDevice);

    //
    dim3 grid(1, 1, 1);
    dim3 block(1, 1, 1);

    add<<<grid, block>>>(da, db, dc, dclock);
    hipDeviceSynchronize();

    hipMemcpy(hc, dc, sizeof(int)*100,hipMemcpyDeviceToHost);
    hipMemcpy(clock, dclock, sizeof(long long int )*1,hipMemcpyDeviceToHost);

    int cnt = 0 ;
    for (int i =0;i<100;++i) {
        printf("i: %d - hc: %d \t", i, hc[i]);
        if (hc[i]==2)
            cnt++;
    }
    printf("\n");

    printf("got %d hc\n", cnt);
    printf("clock: %llu\n", *clock);

    // NOTE: this is incorrect; because within clock64 there can be different SASS
    float avg = static_cast<float>(*clock) / static_cast<float>(cnt);
    printf("average: %f\n", avg);

    return 0;
}
