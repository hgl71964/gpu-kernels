 #include <cstdio>
 #include <vector>
 #include <algorithm>
 
 #include <hipblaslt.h>
 #include <hip/hip_runtime_api.h>
 
 #include "helpers.h"


 float median(std::vector<float>& times) {
     const size_t size = times.size();
     if (size == 0) {
         return 0;
     }
 
     std::sort(times.begin(), times.end());
 
     const size_t mid = size / 2;
     if (size % 2 == 0) {
         return (times[mid] + times[mid - 1]) / 2;
     }
     else {
         return times[mid];
     }
 }
 
 /// Sample wrapper executing single precision gemm algorithm auto tuning by querying cublasLt heuristics for best algorithms,
 /// iterate over the results and pick the algorithm that have the best performance for the given problem
 ///
 /// pointer mode is always host, to change it configure the appropriate matmul descriptor attribute
 /// matmul is not using cublas handle's configuration of math mode, here tensor ops are implicitly allowed; to change
 /// this configure appropriate attribute in the preference handle
 void LtSgemmSimpleAutoTuning(hipblasLtHandle_t ltHandle,
                              hipblasOperation_t transa,
                              hipblasOperation_t transb,
                              int m,
                              int n,
                              int k,
                              const float *alpha, /* host pointer */
                              const float *A,
                              int lda,
                              const float *B,
                              int ldb,
                              const float *beta, /* host pointer */
                              float *C,
                              int ldc,
                              void *workspace,
                              size_t workspaceSize,
                              hipblasLtMatmulAlgo_t& algo) {
     hipblasLtMatmulDesc_t operationDesc = NULL;
     hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL;
     hipblasLtMatmulPreference_t preference = NULL;
 
     const int requestedAlgoCount = 8;
     int returnedResults = 0;
     hipblasLtMatmulHeuristicResult_t heuristicResult[requestedAlgoCount] = { 0 };
     int bestAlgoIdx = 0;
     float time = 0;
     float bestAlgoTime = 0;
     hipStream_t stream;
     hipEvent_t startEvent, stopEvent;
 
     // create operation desciriptor; see hipblasLtMatmulDescAttributes_t for details about defaults; here we just need to
     // set the transforms for A and B
     checkCublasStatus(hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
     checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
     checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));
 
     // create matrix descriptors, we are good with the details here so no need to set any extra attributes
     checkCublasStatus(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_32F, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda));
     checkCublasStatus(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_32F, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb));
     checkCublasStatus(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32F, m, n, ldc));
 
     // create preference handle; here we could use extra attributes to disable tensor ops or to make sure algo selected
     // will work with badly aligned A, B, C; here for simplicity we just assume A,B,C are always well aligned (e.g.
     // directly come from hipMalloc)
     checkCublasStatus(hipblasLtMatmulPreferenceCreate(&preference));
     checkCublasStatus(hipblasLtMatmulPreferenceSetAttribute(preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
         &workspaceSize, sizeof(workspaceSize)));
 
     // we just need the best available heuristic to try and run matmul. There is no guarantee this will work, e.g. if A
     // is badly aligned, you can request more (e.g. 32) algos and try to run them one by one until something works
     checkCublasStatus(hipblasLtMatmulAlgoGetHeuristic(ltHandle, operationDesc, Adesc, Bdesc, Cdesc, Cdesc, preference,
         requestedAlgoCount, heuristicResult, &returnedResults));
 
     if (returnedResults == 0) {
         checkCublasStatus(HIPBLAS_STATUS_NOT_SUPPORTED);
     }
 
     checkCudaStatus(hipStreamCreate(&stream));
     checkCudaStatus(hipEventCreate(&startEvent));
     checkCudaStatus(hipEventCreate(&stopEvent));
 
     constexpr int repeatAlgoCheck = 5;
     std::vector<float> algoTimes(repeatAlgoCheck);
 
     for (int algoIdx = 0; algoIdx < returnedResults; algoIdx++) {
         for (int checkIdx = 0; checkIdx < repeatAlgoCheck; checkIdx++) {
             checkCudaStatus(hipEventRecord(startEvent, stream));
 
             checkCublasStatus(hipblasLtMatmul(ltHandle,
                                             operationDesc,
                                             alpha,
                                             A,
                                             Adesc,
                                             B,
                                             Bdesc,
                                             beta,
                                             C,
                                             Cdesc,
                                             C,
                                             Cdesc,
                                             &heuristicResult[algoIdx].algo,
                                             workspace,
                                             workspaceSize,
                                             stream));
 
             checkCudaStatus(hipEventRecord(stopEvent, stream));
             checkCudaStatus(hipEventSynchronize(stopEvent));
             checkCudaStatus(hipEventElapsedTime(&time, startEvent, stopEvent));
             algoTimes[checkIdx] = time;
         }
 
         time = median(algoTimes);
 
         if (algoIdx == 0 || time < bestAlgoTime) {
             bestAlgoTime = time;
             bestAlgoIdx = algoIdx;
         }
     }
 
     memcpy(&algo, &heuristicResult[bestAlgoIdx].algo, sizeof(algo));
 
     // descriptors are no longer needed as all GPU work was already enqueued
     if (preference) checkCublasStatus(hipblasLtMatmulPreferenceDestroy(preference));
     if (Cdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Cdesc));
     if (Bdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Bdesc));
     if (Adesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Adesc));
     if (operationDesc) hipblasLtMatmulDescDestroy(operationDesc);
     if (stream) checkCudaStatus(hipStreamDestroy(stream));
     if (startEvent) checkCudaStatus(hipEventDestroy(startEvent));
     if (stopEvent) checkCudaStatus(hipEventDestroy(stopEvent));
 }

void printAlgo(const hipblasLtMatmulAlgo_t& algo) {
    int algoId, tile, swizzle, customOption, numSplitsK, reductionScheme;

    checkCublasStatus(cublasLtMatmulAlgoConfigGetAttribute(&algo, CUBLASLT_ALGO_CONFIG_ID, &algoId, sizeof(algoId), NULL));
    checkCublasStatus(cublasLtMatmulAlgoConfigGetAttribute(&algo, CUBLASLT_ALGO_CONFIG_TILE_ID, &tile, sizeof(tile), NULL));
    checkCublasStatus(cublasLtMatmulAlgoConfigGetAttribute(&algo, CUBLASLT_ALGO_CONFIG_SPLITK_NUM, &numSplitsK, sizeof(numSplitsK), NULL));
    checkCublasStatus(cublasLtMatmulAlgoConfigGetAttribute(&algo, CUBLASLT_ALGO_CONFIG_REDUCTION_SCHEME, &reductionScheme, sizeof(reductionScheme), NULL));
    checkCublasStatus(cublasLtMatmulAlgoConfigGetAttribute(&algo, CUBLASLT_ALGO_CONFIG_CTA_SWIZZLING, &swizzle, sizeof(swizzle), NULL));
    checkCublasStatus(cublasLtMatmulAlgoConfigGetAttribute(&algo, CUBLASLT_ALGO_CONFIG_CUSTOM_OPTION, &customOption, sizeof(customOption), NULL));

    printf("algo={ Id=%d, tileIdx=%d splitK=%d reduc=%d swizzle=%d custom=%d }\n",
        algoId, tile, numSplitsK, reductionScheme, swizzle, customOption);
}

int main() {
    TestBench<float> props(1024, 1024, 1024, 2.0f, 0.0f, 1024 * 1024 * 4);

    hipblasLtMatmulAlgo_t algo;

    props.run([&props, &algo] {
        LtSgemmSimpleAutoTuning(props.ltHandle,
                                HIPBLAS_OP_N,
                                HIPBLAS_OP_N,
                                props.m,
                                props.n,
                                props.k,
                                &props.alpha,
                                props.Adev,
                                props.m,
                                props.Bdev,
                                props.k,
                                &props.beta,
                                props.Cdev,
                                props.m,
                                props.workspace,
                                props.workspaceSize,
                                algo);
    });

    printAlgo(algo);

    return 0;
}