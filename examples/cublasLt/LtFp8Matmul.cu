#include "hip/hip_runtime.h"

#include "helpers.h"
#include <hipblaslt.h>

// compile: nvcc -arch sm_89 LtFp8Matmul.cu  -lcublasLt



/// Sample wrapper executing fp8 matmul with hipblasLtMatmul, with addition of per-tensor scaling, amax calculations, and
/// the workspace to support split-K algorithms.
///
/// pointer mode is for alpha and beta is always host, to change it configure the appropriate matmul descriptor
/// attribute matmul is not using cublas handle's configuration of math mode, here tensor ops are implicitly allowed; to
/// change this configure appropriate attribute in the preference handle
void LtFp8Matmul(hipblasLtHandle_t ltHandle,
    int m,
    int n,
    int k,
    const float *alpha, /* host pointer */
    const float *a_scale, /* device pointer */
    const __hip_fp8_e4m3_fnuz *A,
    int lda,
    const float *b_scale, /* device pointer */
    const __hip_fp8_e4m3_fnuz *B,
    int ldb,
    const float *c_scale, /* device pointer */
    __hip_fp8_e4m3_fnuz *D,
    int ldc,
    const float *d_scale, /* device pointer */
    float *amax_d, /* device pointer */
    void *workspace,
    size_t workspaceSize) {

hipblasLtMatmulDesc_t operationDesc = NULL;
hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL, Ddesc = NULL;
hipblasLtMatmulPreference_t preference = NULL;

hipblasOperation_t transa = HIPBLAS_OP_T;
hipblasOperation_t transb = HIPBLAS_OP_N;
float beta = 0.0; // Can be non-zero starting from 12.0

int returnedResults                             = 0;
hipblasLtMatmulHeuristicResult_t heuristicResult = {};

// create operation desciriptor; see hipblasLtMatmulDescAttributes_t for details about defaults; here we just need to
// set the transforms for A and B
checkCublasStatus(hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));

// set scaling factors
checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_A_SCALE_POINTER, &a_scale, sizeof(a_scale)));
checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_B_SCALE_POINTER, &b_scale, sizeof(b_scale)));
checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_C_SCALE_POINTER, &c_scale, sizeof(c_scale)));
checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_D_SCALE_POINTER, &d_scale, sizeof(d_scale)));
checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_AMAX_D_POINTER, &amax_d, sizeof(amax_d)));

// create matrix descriptors, we are good with the details here so no need to set any extra attributes
// table of supported type combinations can be found in the documentation: https://docs.nvidia.com/cuda/cublas/index.html#cublasltmatmul
checkCublasStatus(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_8F_E4M3, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda));
checkCublasStatus(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_8F_E4M3, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb));
checkCublasStatus(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_16BF, m, n, ldc));
checkCublasStatus(hipblasLtMatrixLayoutCreate(&Ddesc, HIP_R_8F_E4M3, m, n, ldc));

// create preference handle; here we could use extra attributes to disable tensor ops or to make sure algo selected
// will work with badly aligned A, B, C; here for simplicity we just assume A,B,C are always well aligned (e.g.
// directly come from hipMalloc)
checkCublasStatus(hipblasLtMatmulPreferenceCreate(&preference));
checkCublasStatus(hipblasLtMatmulPreferenceSetAttribute(preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize)));

// we just need the best available heuristic to try and run matmul. There is no guarantee this will work, e.g. if A
// is badly aligned, you can request more (e.g. 32) algos and try to run them one by one until something works
checkCublasStatus(hipblasLtMatmulAlgoGetHeuristic(ltHandle, operationDesc, Adesc, Bdesc, Cdesc, Ddesc, preference, 1, &heuristicResult, &returnedResults));

if (returnedResults == 0) {
checkCublasStatus(HIPBLAS_STATUS_NOT_SUPPORTED);
}

checkCublasStatus(hipblasLtMatmul(ltHandle,
                        operationDesc,
                        alpha,
                        A,
                        Adesc,
                        B,
                        Bdesc,
                        &beta,
                        nullptr,
                        Cdesc,
                        D,
                        Ddesc,
                        &heuristicResult.algo,
                        workspace,
                        workspaceSize,
                        0));

// descriptors are no longer needed as all GPU work was already enqueued
if (preference) checkCublasStatus(hipblasLtMatmulPreferenceDestroy(preference));
if (Ddesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Ddesc));
if (Cdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Cdesc));
if (Bdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Bdesc));
if (Adesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Adesc));
if (operationDesc) checkCublasStatus(hipblasLtMatmulDescDestroy(operationDesc));

}

int main() {
    TestBench<__hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, float> props(64, 128, 256, 2.0f, 0.0f /* ignored */, 32ULL * 1024 * 1024);

    props.run([&props] {
        LtFp8Matmul(props.ltHandle,
                    props.m,
                    props.n,
                    props.k,
                    &props.alpha,
                    props.AscaleDev,
                    props.Adev,
                    props.k,
                    props.BscaleDev,
                    props.Bdev,
                    props.k,
                    props.CscaleDev,
                    props.Cdev,
                    props.m,
                    props.DscaleDev,
                    props.DamaxDev,
                    props.workspace,
                    props.workspaceSize);
    });

    return 0;
}