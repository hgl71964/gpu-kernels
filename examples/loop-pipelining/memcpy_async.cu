#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"

#include <cstdlib>  // for rand
#include <ctime> // for time()


// c++ interface for async global -> shared
// ptx: cp.async
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>




#define cudaErrCheck(stat)                         \
    {                                              \
        cudaErrCheck_((stat), __FILE__, __LINE__); \
    }
void cudaErrCheck_(hipError_t stat, const char* file, int line)
{
    if (stat != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
    }
}

int cdiv(const int length, const int block_size) {
    return (length + block_size - 1)/block_size;
}

__global__ void tile_mm_sync(float *A, float *B, float *C, const int M, const int N, const int K) {

    extern __shared__ float shared_storage[];
    float *sa = reinterpret_cast<float *>(shared_storage);
    float *sb = sa + 4 * 4;

    int block_k = 4;

    int row_start = blockIdx.x * blockDim.x;
    int col_start = blockIdx.y * blockDim.y;


    float accu = 0.0f;
    for (int k = 0; k < K; k+=block_k) {
        sa[threadIdx.x*4 + threadIdx.y] = A[(row_start+threadIdx.x)*K+threadIdx.y + k];
        sb[threadIdx.x*4 + threadIdx.y] = B[(threadIdx.x+k)*N+col_start+threadIdx.y];
        __syncthreads();

        // accu
        for (int kk = 0; kk < block_k; ++kk) {
            //accu += sa[threadIdx.x][kk] * sb[kk][threadIdx.y];
            accu += sa[threadIdx.x*4 + kk] * sb[kk*4 + threadIdx.y];
        }
        __syncthreads();
    }

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < M && col < N) {
        C[row*N+col] = accu;
    }
}

__global__ void tile_mm_async(float *A, float *B, float *C, const int M, const int N, const int K) {
    // https://docs.nvidia.com/cuda/cuda-c-programming-guide/#asynchronous-data-copies
    // https://forums.developer.nvidia.com/t/using-memcpy-async-in-matrix-transpose/281051

    extern __shared__ float shared_storage[];
    float *sa = reinterpret_cast<float *>(shared_storage);
    float *sb = sa + 4 * 4;

    int block_k = 4;
    int row_start = blockIdx.x * blockDim.x;
    int col_start = blockIdx.y * blockDim.y;

    auto TB = cooperative_groups::this_thread_block();
    // if (blockIdx.x == 0 && blockIdx.y == 0) {
    //     // thread rank is a flattened index of threadIdx
    //     printf("block size: %d, thread rank: %d, thread idx: %d, %d\n", TB.size(), TB.thread_rank(), threadIdx.x, threadIdx.y);
    // }

    float accu = 0.0f;
    for (int k = 0; k < K; k+=block_k) {

        //SYNC copy (each thread map to a element)
        // sa[threadIdx.x*4 + threadIdx.y] = A[(row_start+threadIdx.x)*K+threadIdx.y + k];
        // sb[threadIdx.x*4 + threadIdx.y] = B[(threadIdx.x+k)*N+col_start+threadIdx.y];
        // __syncthreads();

        // C++ ASYNC COPY 
        // NOTE: (it is collective, so all thread in a TB must call with the same arguments, otherwise it is undefined behavior)
        cooperative_groups::memcpy_async(TB, sa, A+row_start*K+k, sizeof(float)*4);
        cooperative_groups::memcpy_async(TB, sa+4, A+(row_start+1)*K+k, sizeof(float)*4);
        cooperative_groups::memcpy_async(TB, sa+8, A+(row_start+2)*K+k, sizeof(float)*4);
        cooperative_groups::memcpy_async(TB, sa+12, A+(row_start+3)*K+k, sizeof(float)*4);

        cooperative_groups::memcpy_async(TB, sb, B+(k)*N + col_start, sizeof(float)*4);
        cooperative_groups::memcpy_async(TB, sb+4, B+(k+1)*N + col_start, sizeof(float)*4);
        cooperative_groups::memcpy_async(TB, sb+8, B+(k+2)*N + col_start, sizeof(float)*4);
        cooperative_groups::memcpy_async(TB, sb+12, B+(k+3)*N + col_start, sizeof(float)*4);

        cooperative_groups::wait(TB); // Wait for all copies to complete

        // accu
        for (int kk = 0; kk < block_k; ++kk) {
            //accu += sa[threadIdx.x][kk] * sb[kk][threadIdx.y];
            accu += sa[threadIdx.x*4 + kk] * sb[kk*4 + threadIdx.y];
        }
        __syncthreads();
    }

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < M && col < N) {
        C[row*N+col] = accu;
    }
}


int main() {
    srand(time(0));

    const int M = 64;
    const int N = 64;
    const int K = 64;

    const int block_m = 4;
    const int block_n = 4;

    float *ha = (float *)malloc(sizeof(float)*M*K);
    float *hb = (float *)malloc(sizeof(float)*N*K);
    float *hc = (float *)malloc(sizeof(float)*M*N);
    float *ref_c = (float *)malloc(sizeof(float)*M*N);

    float *da, *db, *dc;
    hipMalloc((void**)&db, sizeof(float)*N*K);
    hipMalloc((void **)&da, sizeof(float)*M*K);
    hipMalloc((void**)&dc, sizeof(float)*M*N);

	float min = 0.0f;
	float max = 1.0f;
    for (int i =0;i<M*K;++i) {
        ha[i] = min + static_cast<float>(rand()) / (static_cast<float>(RAND_MAX / (max - min)));
    }
    for (int i =0;i<N*K;++i) {
        hb[i] = min + static_cast<float>(rand()) / (static_cast<float>(RAND_MAX / (max - min)));
    }
    for (int i = 0;i<M*N;++i) {
        hc[i] = 1.0f;
    }
    for (int i =0;i<M;++i) {
        for (int j =0;j<N;++j){
            float accu = 0.0f;

            //assume both row-major
            for (int k = 0; k < K; ++k) {
                accu += ha[i*K+k] * hb[k*N+j];
            }
            ref_c[i*N+j] = accu;
        }
    }
    hipMemcpy(da, ha, sizeof(float)*M*K,hipMemcpyHostToDevice);
    hipMemcpy(db, hb, sizeof(float)*N*K, hipMemcpyHostToDevice);
    hipMemcpy(dc, hc, sizeof(float)*M*N, hipMemcpyHostToDevice);


    dim3 block(block_m, block_n, 1);
    dim3 grid(cdiv(M, block_m), cdiv(N, block_n), 1);

        std::cout << "GEMM: " << M << "; " << N << "; " << K << std::endl;
        std::cout << "grid: " << grid.x << "; " << grid.y << std::endl;
        std::cout << "block: " << block.x << "; " << block.y << std::endl;

    size_t smem = 2*block_m*block_n*4; // num of bytes
    // tile_mm_sync<<<grid, block, smem, nullptr>>>(da, db, dc, M, N, K);
     tile_mm_async<<<grid, block, smem, nullptr>>>(da, db, dc, M, N, K);

    hipMemcpy(hc, dc, sizeof(float)*N*M,hipMemcpyDeviceToHost);
    cudaErrCheck(hipDeviceSynchronize());


        int err=0;
        for (int i = 0; i < M; ++i) {
                for (int j = 0; j < N; ++j) {
			if (std::abs(hc[i*N+j] - ref_c[i*N+j]) > 1e-4) {
			    std::cout << i << "." << j << ": " << hc[i*N+j] << " " << ref_c[i*N+j] << std::endl;
                            err++;
			}
                        if (err > 10)
                                break;
                }
                if (err > 10)
                        break;
        }

        if (err > 0) 
            printf("MM check has %d errors!\n", err);
        else
            printf("OK\n");

}