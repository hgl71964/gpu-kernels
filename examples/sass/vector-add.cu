
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// compile with nvcc
// dump sass: cuobjdump --dump-sass vector-add.o

__global__ void vecAdd(double *a, double *b, double *c, int n)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;

    // Make sure we do not go out of bounds
    if (id < n)
        c[id] = a[id] + b[id];
}

int main( int argc, char* argv[] )
{
    // Size of vectors
    int n = 100000;

    double *h_a;
    double *h_b;
    double *h_c;

    double *d_a;
    double *d_b;
    double *d_c;

    size_t bytes = n*sizeof(double);

    h_a = (double*)malloc(bytes);
    h_b = (double*)malloc(bytes);
    h_c = (double*)malloc(bytes);

    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    int i;
    for( i = 0; i < n; i++ ) {
        h_a[i] = sin(i)*sin(i);
        h_b[i] = cos(i)*cos(i);
    }

    // Copy host vectors to device
    hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);

    int blockSize, gridSize;

    blockSize = 1024;

    gridSize = (int)ceil((float)n/blockSize);

    vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

    hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );

    double sum = 0;
    for(i=0; i<n; i++)
        sum += h_c[i];
    printf("final result: %f\n", sum/n);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
