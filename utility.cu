
#include "hip/hip_runtime.h"

#include <iostream>
#include <stdio.h>

using namespace std;

__global__ void hello()
{
    // __shared__ char smem;
    extern __shared__ uint8_t smem[];  // dynamic shared mem

    // when running insides docker container, device cannot print anything
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello World from device!\n");
    printf("device idx: %d \n", idx);
}

// .cu file can be directly compiled by cmake's CUDA
int main()
{
    dim3 block;
    block.x = 2;
    // block.y = 1;

    auto kernel = hello;

    uint32_t smem_size = 4;

    const uint32_t num_warps = 4;
    const uint32_t warp_size = 32;

    int dev_id = 0;

    constexpr uint32_t num_threads = num_warps * warp_size;

    hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize, smem_size);

    int num_sm = 0;
    hipDeviceGetAttribute(&num_sm, hipDeviceAttributeMultiprocessorCount, dev_id);

    int num_blocks_per_sm = 0;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks_per_sm, kernel, num_threads, smem_size);

    cout << "num_sm: " << num_sm << " num_blocks_per_sm: " << num_blocks_per_sm << endl;


    hello<<<1, block>>>();
    hipDeviceSynchronize();

    int driver_version;
    int runtime_version;
    hipRuntimeGetVersion(&runtime_version);
    hipDriverGetVersion(&driver_version);
    cout << "driver: " << driver_version << endl;
    cout << "runtime: " << runtime_version << endl;

    return 0;
}
